#include <random>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <iostream>

#define M 4096
#define N 4096
#define K 4096
#define THREAD_NUM 1024

void mat_init (int *mat_a, int *mat_b, int *mat_res, int m, int n, int k) {
    srand(0);
    // mat_a
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < n; j++) {
            mat_a[i * m + j] = random() % 10;
        }
    }
    // mat_b
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < k; j++) {
            mat_b[i * n + j] = random() % 10;
        }
    }
    // mat_res
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            mat_res[i * m + j] = 0;
        }
    }
}

void cpu_mat_mul (int *mat_a, int *mat_b, int *mat_res, int m, int n, int k) {
    for (int i = 0; i < m; i++) {
        for (int j = 0; j < k; j++) {
            int tmp = 0;
            for (int q = 0; q < n; q++) {
                tmp += mat_a[i * n + q] * mat_b[q * k + j];
            }
            mat_res[i * m + j] = tmp;
        }
    }
}

__global__ void gpu_mat_mul (int *mat_a, int *mat_b, int *mat_res, int m, int n, int k) {
    int tid = threadIdx.x;
    int bid = blockIdx.x;
    int idx = bid * THREAD_NUM + tid;

    int row = idx / m;
    int col = idx % m;

    if (row < m && col < n) {
        int tmp = 0;
        for (int q = 0; q < n; q++) {
            tmp += mat_a[row * n + q] * mat_b[q * k + col];
        }
        mat_res[row * m + col] = tmp;
    }
}

void print_mat(int *mat, int row, int col) {
    std::cout << std::endl;
    for (int i = 0; i < row; i++) {
        for (int j = 0; j < col; j++) {
            std::cout << mat[i * col + j] <<" ";
        }
        std::cout << std::endl;
    }
    std::cout << std::endl;
}

int main() {
    int *a, *b, *res;
    int m = M, n = N, k = K;

    // 分配CPU内存
    a = (int *)malloc(m * n * sizeof(int));
    b = (int *)malloc(n * k * sizeof(int));
    res = (int *)malloc(m * k * sizeof(int));
    // 矩阵初始化
    mat_init(a, b, res, m, n, k);
    // print_mat(a, m, n);
    // print_mat(b, n, k);

    struct timeval cpu_start, cpu_end, gpu__start, gpu_end;
    double cpu_time, gpu_time;

    // cpu_mat_mul
    gettimeofday(&cpu_start, nullptr);
    cpu_mat_mul(a, b, res, m, n, k);
    gettimeofday(&cpu_end, nullptr);
    cpu_time = (cpu_end.tv_sec*1000000 + cpu_end.tv_usec) - (cpu_start.tv_sec*1000000 + cpu_start.tv_usec); //um

    // 打印结果
    // print_mat(res, m, k);

    // gpu_mat_mul
    //从主机拷贝数据到设备
    int *dev_a, *dev_b, *dev_res;
    hipMalloc((void **)&dev_a, m * n * sizeof(int));
    hipMalloc((void **)&dev_b, n * k * sizeof(int));
    hipMalloc((void **)&dev_res, m * k * sizeof(int));
    hipMemcpy(dev_a, a, m * n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, n * k * sizeof(int), hipMemcpyHostToDevice);

    int block_num = (m * n - 1) / THREAD_NUM + 1;

    gettimeofday(&gpu__start, nullptr);
    gpu_mat_mul<<<block_num, THREAD_NUM>>>(dev_a, dev_b,dev_res, m, n, k);
    hipMemcpy(res, dev_res, m * k * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    gettimeofday(&gpu_end, nullptr);
    gpu_time = (gpu_end.tv_sec*1000000 + gpu_end.tv_usec) - (gpu__start.tv_sec*1000000 + gpu__start.tv_usec); //um

    // 打印结果
    // print_mat(res, m, k);

    // 输出结果
    std::cout << "矩阵规模：[" << m  <<"][" << n << "] * [" << n << "][" << k << "]\n";
    std::cout << "cpu_time = " << cpu_time/1000000 << " s\n" << "gpu_time = " << gpu_time/1000000 << " s\n" << "加速比 = " << cpu_time/gpu_time << std::endl;

    free(a);
    free(b);
    free(res);
    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_res);

    return 0;
}